#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "plugin.h"
#include "hip/hip_fp16.h"
#include "gatherNMSOutputs.h"
#include <array>

namespace
{
// __half minus with fallback to float for old sm
inline __device__ __half minus_fb(const __half & a, const __half & b) {
#if __CUDA_ARCH__ >= 530
    return a - b;
#else
    return __float2half(__half2float(a) - __half2float(b));
#endif
}

// overload for float
inline __device__ float minus_fb(const float & a, const float & b) {
    return a - b;
}

template <typename T_BBOX>
__device__ T_BBOX saturate(T_BBOX v)
{
    return max(min(v, T_BBOX(1)), T_BBOX(0));
}

template <>
__device__ __half saturate(__half v)
{
#if __CUDA_ARCH__ >= 800
    return __hmax(__hmin(v, __half(1)), __half(0));
#elif __CUDA_ARCH__ >= 530
    return __hge(v, __half(1)) ? __half(1) : (__hle(v, __half(0)) ? __half(0) : v);
#else
    return max(min(v, float(1)), float(0));
#endif
}
}

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSOutputs_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        const T_BBOX* ldmkData,
        int* numDetections,
        T_BBOX* nmsedBoxes,
        T_BBOX* nmsedScores,
        // T_BBOX* nmsedClasses,
        T_BBOX* nmsedLdmks,
        bool clipBoxes,
        const T_SCORE scoreShift
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            // nmsedClasses[i] = -1;
            nmsedScores[i] = 0;
            nmsedBoxes[i * 4] = 0;
            nmsedBoxes[i * 4 + 1] = 0;
            nmsedBoxes[i * 4 + 2] = 0;
            nmsedBoxes[i * 4 + 3] = 0;

            nmsedLdmks[i * 10] = 0;
            nmsedLdmks[i * 10 + 1] = 0;
            nmsedLdmks[i * 10 + 2] = 0;
            nmsedLdmks[i * 10 + 3] = 0;
            nmsedLdmks[i * 10 + 4] = 0;
            nmsedLdmks[i * 10 + 5] = 0;
            nmsedLdmks[i * 10 + 6] = 0;
            nmsedLdmks[i * 10 + 7] = 0;
            nmsedLdmks[i * 10 + 8] = 0;
            nmsedLdmks[i * 10 + 9] = 0;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset) * 4;
            // nmsedClasses[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedScores[i] = score;                                                        // confidence score
            nmsedScores[i] = minus_fb(nmsedScores[i], scoreShift);
            const T_BBOX xMin = bboxData[bboxId];
            const T_BBOX yMin = bboxData[bboxId + 1];
            const T_BBOX xMax = bboxData[bboxId + 2];
            const T_BBOX yMax = bboxData[bboxId + 3];
            // clipped bbox xmin
            nmsedBoxes[i * 4] = clipBoxes ? saturate(xMin) : xMin;
            // clipped bbox ymin
            nmsedBoxes[i * 4 + 1] = clipBoxes ? saturate(yMin) : yMin;
            // clipped bbox xmax
            nmsedBoxes[i * 4 + 2] = clipBoxes ? saturate(xMax) : xMax;
            // clipped bbox ymax
            nmsedBoxes[i * 4 + 3] = clipBoxes ? saturate(yMax) : yMax;
            const int ldmkOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int ldmkId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + ldmkOffset) * 10;
            nmsedLdmks[i * 10] = saturate(ldmkData[ldmkId]);
            nmsedLdmks[i * 10 + 1] = saturate(ldmkData[ldmkId+1]);
            nmsedLdmks[i * 10 + 2] = saturate(ldmkData[ldmkId+2]);
            nmsedLdmks[i * 10 + 3] = saturate(ldmkData[ldmkId+3]);
            nmsedLdmks[i * 10 + 4] = saturate(ldmkData[ldmkId+4]);
            nmsedLdmks[i * 10 + 5] = saturate(ldmkData[ldmkId+5]);
            nmsedLdmks[i * 10 + 6] = saturate(ldmkData[ldmkId+6]);
            nmsedLdmks[i * 10 + 7] = saturate(ldmkData[ldmkId+7]);
            nmsedLdmks[i * 10 + 8] = saturate(ldmkData[ldmkId+8]);
            nmsedLdmks[i * 10 + 9] = saturate(ldmkData[ldmkId+9]);

            atomicAdd(&numDetections[i / keepTopK], 1);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    const void* ldmkData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    // void* nmsedClasses,
    void* nmsedLdmks,
    bool clipBoxes,
    const float scoreShift
    )
{
    hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
                                                                           (T_BBOX*) ldmkData,
                                                                           (int*) numDetections,
                                                                           (T_BBOX*) nmsedBoxes,
                                                                           (T_BBOX*) nmsedScores,
                                                                        //    (T_BBOX*) nmsedClasses,
                                                                           (T_BBOX*) nmsedLdmks,
                                                                           clipBoxes,
                                                                           T_SCORE(scoreShift)
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               const void*,
                               void*,
                               void*,
                               void*,
                            //    void*,
                               void*,
                               bool,
                               const float);
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::array<nmsOutLaunchConfig, 2> nmsOutLCOptions = {
  nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSOutputs_gpu<float, float>),
  nmsOutLaunchConfig(DataType::kHALF, DataType::kHALF, gatherNMSOutputs_gpu<__half, __half>)
};

pluginStatus_t gatherNMSOutputs(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DataType DT_BBOX,
    const DataType DT_SCORE,
    // const DataType DT_LDMK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    const void* ldmkData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    // void* nmsedClasses,
    void* nmsedLdmks,
    bool clipBoxes,
    const float scoreShift
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutLCOptions.size(); ++i)
    {
        if (lc == nmsOutLCOptions[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
            return nmsOutLCOptions[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          ldmkData,
                                          numDetections,
                                          nmsedBoxes,
                                          nmsedScores,
                                        //   nmsedClasses,
                                          nmsedLdmks,
                                          clipBoxes,
                                          scoreShift
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}
